#include "hip/hip_runtime.h"
// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/Plasticity/InhibitorySTDPPlasticity.hpp"

SPIKE_EXPORT_BACKEND_TYPE(CUDA, InhibitorySTDPPlasticity);

namespace Backend {
  namespace CUDA {
    InhibitorySTDPPlasticity::~InhibitorySTDPPlasticity() {
      CudaSafeCall(hipFree(vogels_pre_memory_trace));
      CudaSafeCall(hipFree(vogels_post_memory_trace));
      if (vogels_memory_trace_reset){
        free(vogels_memory_trace_reset);
      }
    }

    void InhibitorySTDPPlasticity::reset_state() {
      STDPPlasticity::reset_state();

      CudaSafeCall(hipMemcpy((void*)vogels_pre_memory_trace,
                              (void*)vogels_memory_trace_reset,
                              sizeof(float)*total_number_of_plastic_synapses,
                              hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy((void*)vogels_post_memory_trace,
                              (void*)vogels_memory_trace_reset,
                              sizeof(float)*total_number_of_plastic_synapses,
                              hipMemcpyHostToDevice));
    }

    void InhibitorySTDPPlasticity::prepare() {
      STDPPlasticity::prepare();

      vogels_memory_trace_reset = (float*)malloc(sizeof(float)*total_number_of_plastic_synapses);
      for (int i=0; i < total_number_of_plastic_synapses; i++){
        vogels_memory_trace_reset[i] = 0.0f;
      }

      allocate_device_pointers();
    }

    void InhibitorySTDPPlasticity::allocate_device_pointers() {
      // The following doesn't do anything in original code...
      // ::Backend::CUDA::STDPPlasticity::allocate_device_pointers();

      CudaSafeCall(hipMalloc((void **)&vogels_pre_memory_trace, sizeof(int)*total_number_of_plastic_synapses));
      CudaSafeCall(hipMalloc((void **)&vogels_post_memory_trace, sizeof(int)*total_number_of_plastic_synapses));
    }

    void InhibitorySTDPPlasticity::apply_stdp_to_synapse_weights(unsigned int current_time_in_timesteps, float timestep) {

    // Vogels update rule requires a neuron wise memory trace. This must be updated upon neuron firing.
    vogels_apply_stdp_to_synapse_weights_kernel<<<neurons_backend->number_of_neuron_blocks_per_grid, neurons_backend->threads_per_block>>>
       (synapses_backend->postsynaptic_neuron_indices,
       synapses_backend->presynaptic_neuron_indices,
       synapses_backend->delays,
       neurons_backend->d_neuron_data,
       input_neurons_backend->d_neuron_data,
       synapses_backend->synaptic_efficacies_or_weights,
       vogels_pre_memory_trace,
       vogels_post_memory_trace,
       //frontend()->stdp_params->tau_istdp,
       expf(- timestep / frontend()->stdp_params->tau_istdp),
       *(frontend()->stdp_params),
       timestep,
       frontend()->model->timestep_grouping,
       current_time_in_timesteps,
       plastic_synapse_indices,
       total_number_of_plastic_synapses);
    CudaCheckError();
    }

    __global__ void vogels_apply_stdp_to_synapse_weights_kernel
          (int* d_postsyns,
           int* d_presyns,
           int* d_syndelays,
           spiking_neurons_data_struct* neuron_data,
           spiking_neurons_data_struct* input_neuron_data,
           float* d_synaptic_efficacies_or_weights,
           float* vogels_pre_memory_trace,
           float* vogels_post_memory_trace,
           float trace_decay,
           struct inhibitory_stdp_plasticity_parameters_struct stdp_vars,
           float timestep,
           int timestep_grouping,
           unsigned int current_time_in_timesteps,
           int* d_plastic_synapse_indices,
           size_t total_number_of_plastic_synapses){
      // Global Index
      int indx = threadIdx.x + blockIdx.x * blockDim.x;

      // Running though all neurons
      while (indx < total_number_of_plastic_synapses) {
        int idx = d_plastic_synapse_indices[indx];

        // Getting synapse details
        float vogels_pre_memory_trace_val = vogels_pre_memory_trace[indx];
        float vogels_post_memory_trace_val = vogels_post_memory_trace[indx];
        int postid = d_postsyns[idx];
        int preid = d_presyns[idx];
        int bufsize = input_neuron_data->neuron_spike_time_bitbuffer_bytesize[0];
        float old_synaptic_weight = d_synaptic_efficacies_or_weights[idx];
        float new_synaptic_weight = old_synaptic_weight;

        // Correcting for input vs output neuron types
        bool is_input = PRESYNAPTIC_IS_INPUT(preid);
        int corr_preid = CORRECTED_PRESYNAPTIC_ID(preid, is_input);
        uint8_t* pre_bitbuffer = is_input ? input_neuron_data->neuron_spike_time_bitbuffer : neuron_data->neuron_spike_time_bitbuffer;

        /*
        float* pre_last_spike_times = is_input ? input_neuron_data->last_spike_time_of_each_neuron : neuron_data->last_spike_time_of_each_neuron;
        //int pre_spike_g = -1;
        int pre_spike_g = ((int)roundf(pre_last_spike_times[corr_preid] / timestep) - current_time_in_timesteps);
        int post_spike_g = ((int)roundf(neuron_data->last_spike_time_of_each_neuron[postid] / timestep) - current_time_in_timesteps);
        if (pre_spike_g >= timestep_grouping)
          pre_spike_g *= -1;

        vogels_post_memory_trace_val *= expf(-(timestep_grouping*timestep) / trace_decay);
        vogels_pre_memory_trace_val *= expf(-(timestep_grouping*timestep) / trace_decay);

        // Change this if nearest only
        vogels_post_memory_trace_val += (post_spike_g >= 0) ? expf(-((timestep_grouping - post_spike_g - 1)*timestep) / trace_decay) : 0.0f;
        vogels_pre_memory_trace_val += (pre_spike_g >= 0) ? expf(-((timestep_grouping - pre_spike_g - 1)*timestep) / trace_decay) : 0.0f;
          
        float syn_update_val = 0.0f; 
        //old_synaptic_weight = new_synaptic_weight;
        // OnPre Weight Update
        if (pre_spike_g >= 0){
          float temp_post_trace = vogels_post_memory_trace_val;
          temp_post_trace += (post_spike_g > pre_spike_g) ? -expf(-((timestep_grouping - post_spike_g - 1)*timestep) / trace_decay): 0.0f;
          temp_post_trace *= (1.0f / (expf(-(timestep_grouping - pre_spike_g - 1)*timestep / trace_decay))); 
          syn_update_val += stdp_vars.learningrate*(temp_post_trace);
          syn_update_val += - stdp_vars.learningrate*(2.0*stdp_vars.targetrate*stdp_vars.tau_istdp);
          //syn_update_val -= stdp_vars.lambda * stdp_vars.alpha * old_synaptic_weight * temp_post_trace;
        }
        // OnPost Weight Update
        if (post_spike_g >= 0){
          float temp_pre_trace = vogels_pre_memory_trace_val;
          temp_pre_trace += (pre_spike_g > post_spike_g) ? -expf(-((timestep_grouping - pre_spike_g - 1)*timestep) / trace_decay): 0.0f;
          temp_pre_trace *= (1.0f / (expf(-(timestep_grouping - post_spike_g - 1)*timestep / trace_decay))); 
          syn_update_val += stdp_vars.learningrate*(temp_pre_trace);
          //syn_update_val += stdp_vars.lambda * (stdp_vars.w_max - old_synaptic_weight) * temp_pre_trace;
        }

        new_synaptic_weight = old_synaptic_weight + syn_update_val;
        if (new_synaptic_weight < 0.0f)
          new_synaptic_weight = 0.0f;
        
        // Weight Update
        d_synaptic_efficacies_or_weights[idx] = new_synaptic_weight;

        // Correctly set the trace values
        vogels_pre_memory_trace[indx] = vogels_pre_memory_trace_val;
        vogels_post_memory_trace[indx] = vogels_post_memory_trace_val;

        indx += blockDim.x * gridDim.x;
        */
        // Looping over timesteps
        for (int g=0; g < timestep_grouping; g++){	
          // Decaying STDP traces
          vogels_pre_memory_trace_val *= trace_decay;
          vogels_post_memory_trace_val *= trace_decay;
          
          // Bit Indexing to detect spikes
          int postbitloc = (current_time_in_timesteps + g) % (bufsize*8);
          int prebitloc = postbitloc - d_syndelays[idx];
          prebitloc = (prebitloc < 0) ? (bufsize*8 + prebitloc) : prebitloc;

          // OnPre Trace Update
          if (pre_bitbuffer[corr_preid*bufsize + (prebitloc / 8)] & (1 << (prebitloc % 8))){
            vogels_pre_memory_trace_val += 1.0f;
          }
          // OnPost Trace Update
          if (neuron_data->neuron_spike_time_bitbuffer[postid*bufsize + (postbitloc / 8)] & (1 << (postbitloc % 8))){
            vogels_post_memory_trace_val += 1.0f;
          }

          float syn_update_val = 0.0f; 
          old_synaptic_weight = new_synaptic_weight;
          // OnPre Weight Update
          if (pre_bitbuffer[corr_preid*bufsize + (prebitloc / 8)] & (1 << (prebitloc % 8))){
            syn_update_val += stdp_vars.learningrate*(vogels_post_memory_trace_val);
            syn_update_val += -stdp_vars.learningrate*(2.0*stdp_vars.targetrate*stdp_vars.tau_istdp);
          }
          // OnPost Weight Update
          if (neuron_data->neuron_spike_time_bitbuffer[postid*bufsize + (postbitloc / 8)] & (1 << (postbitloc % 8))){
            syn_update_val += stdp_vars.learningrate*(vogels_pre_memory_trace_val);
          }

          new_synaptic_weight = old_synaptic_weight + syn_update_val;
          if (new_synaptic_weight < 0.0f)
            new_synaptic_weight = 0.0f;
        }

        if (new_synaptic_weight > stdp_vars.w_max)
          new_synaptic_weight = stdp_vars.w_max;

        // Weight Update
        d_synaptic_efficacies_or_weights[idx] = new_synaptic_weight;
        
        // Correctly set the trace values
        vogels_pre_memory_trace[indx] = vogels_pre_memory_trace_val;
        vogels_post_memory_trace[indx] = vogels_post_memory_trace_val;

        indx += blockDim.x * gridDim.x;
      }

    }
  }
}
