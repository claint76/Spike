// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/Neurons/PatternedPoissonInputSpikingNeurons.hpp"

SPIKE_EXPORT_BACKEND_TYPE(CUDA, PatternedPoissonInputSpikingNeurons);

namespace Backend {
  namespace CUDA {
    PatternedPoissonInputSpikingNeurons::~PatternedPoissonInputSpikingNeurons() {
      CudaSafeCall(hipFree(stimuli_rates));
    }

    void PatternedPoissonInputSpikingNeurons::allocate_device_pointers() {
      CudaSafeCall(hipMalloc((void **)&stimuli_rates, sizeof(float)*frontend()->total_number_of_rates));
    }

    void PatternedPoissonInputSpikingNeurons::copy_rates_to_device() {
      CudaSafeCall(hipMemcpy(stimuli_rates, frontend()->stimuli_rates, sizeof(float)*frontend()->total_number_of_rates, hipMemcpyHostToDevice));
    }

    void PatternedPoissonInputSpikingNeurons::reset_state() {
      PoissonInputSpikingNeurons::reset_state();
    }

    void PatternedPoissonInputSpikingNeurons::prepare() {
      PoissonInputSpikingNeurons::prepare();
      allocate_device_pointers();
      copy_rates_to_device();
    }

    void PatternedPoissonInputSpikingNeurons::state_update(unsigned int current_time_in_timesteps, float timestep) {
      ::Backend::CUDA::SpikingSynapses* synapses_backend =
        dynamic_cast<::Backend::CUDA::SpikingSynapses*>(frontend()->model->spiking_synapses->backend());
      poisson_update_membrane_potentials_kernel<<<random_state_manager_backend->block_dimensions, random_state_manager_backend->threads_per_block>>>(
         synapses_backend->host_syn_activation_kernel,
         synapses_backend->d_synaptic_data,
         d_neuron_data,
         random_state_manager_backend->states,
         stimuli_rates,
         active,
         membrane_potentials_v,
         timestep,
         frontend()->model->timestep_grouping,
         thresholds_for_action_potential_spikes,
         resting_potentials_v0,
         next_spike_time_of_each_neuron,
         current_time_in_timesteps,
         frontend()->total_number_of_neurons,
         frontend()->current_stimulus_index);

      CudaCheckError();
    }
  }
}
