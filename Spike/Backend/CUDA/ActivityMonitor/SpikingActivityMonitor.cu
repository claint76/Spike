#include "hip/hip_runtime.h"
// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/ActivityMonitor/SpikingActivityMonitor.hpp"

SPIKE_EXPORT_BACKEND_TYPE(CUDA, SpikingActivityMonitor);

namespace Backend {
  namespace CUDA {
    SpikingActivityMonitor::~SpikingActivityMonitor() {
      CudaSafeCall(hipFree(neuron_ids_of_stored_spikes_on_device));
      CudaSafeCall(hipFree(total_number_of_spikes_stored_on_device));
      CudaSafeCall(hipFree(time_in_seconds_of_stored_spikes_on_device));
    }

    void SpikingActivityMonitor::reset_state() {
      ActivityMonitor::reset_state();

      CudaSafeCall(hipMemset(&(total_number_of_spikes_stored_on_device[0]), 0, sizeof(int)));
      CudaSafeCall(hipMemcpy(neuron_ids_of_stored_spikes_on_device, frontend()->reset_neuron_ids, sizeof(int)*frontend()->size_of_device_spike_store, hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy(time_in_seconds_of_stored_spikes_on_device, frontend()->reset_neuron_times, sizeof(float)*frontend()->size_of_device_spike_store, hipMemcpyHostToDevice));
    }

    void SpikingActivityMonitor::prepare() {
      neurons_frontend = frontend()->neurons;
      neurons_backend =
        dynamic_cast<::Backend::CUDA::SpikingNeurons*>(neurons_frontend->backend());
      ActivityMonitor::prepare();

      CudaSafeCall(hipMalloc((void **)&neuron_ids_of_stored_spikes_on_device, sizeof(int)*frontend()->size_of_device_spike_store));
      CudaSafeCall(hipMalloc((void **)&time_in_seconds_of_stored_spikes_on_device, sizeof(float)*frontend()->size_of_device_spike_store));
      CudaSafeCall(hipMalloc((void **)&total_number_of_spikes_stored_on_device, sizeof(int)));
    
      reset_state();
    }
   
    void SpikingActivityMonitor::copy_spikecount_to_front(){
      CudaSafeCall(hipMemcpy((void*)&(frontend()->total_number_of_spikes_stored_on_device[0]), 
                              total_number_of_spikes_stored_on_device, 
                              sizeof(int), hipMemcpyDeviceToHost));
    }

    void SpikingActivityMonitor::copy_spikes_to_front() {
      CudaSafeCall(hipMemcpy((void*)&frontend()->neuron_ids_of_stored_spikes_on_host[frontend()->total_number_of_spikes_stored_on_host], 
                              neuron_ids_of_stored_spikes_on_device, 
                              (sizeof(int)*frontend()->total_number_of_spikes_stored_on_device[0]), 
                              hipMemcpyDeviceToHost));
      CudaSafeCall(hipMemcpy((void*)&frontend()->spike_times_of_stored_spikes_on_host[frontend()->total_number_of_spikes_stored_on_host], 
                              time_in_seconds_of_stored_spikes_on_device, 
                              sizeof(float)*frontend()->total_number_of_spikes_stored_on_device[0], 
                              hipMemcpyDeviceToHost));
    }

    void SpikingActivityMonitor::collect_spikes_for_timestep
    (unsigned int current_time_in_timesteps, float timestep) {
      collect_spikes_for_timestep_kernel<<<neurons_backend->number_of_neuron_blocks_per_grid, neurons_backend->threads_per_block>>>
        (neurons_backend->d_neuron_data,
         total_number_of_spikes_stored_on_device,
         neuron_ids_of_stored_spikes_on_device,
         time_in_seconds_of_stored_spikes_on_device,
         frontend()->model->timestep_grouping,
         current_time_in_timesteps,
         timestep,
         neurons_frontend->total_number_of_neurons);

      CudaCheckError();
    }


    // Collect Spikes
    __global__ void collect_spikes_for_timestep_kernel
    (spiking_neurons_data_struct* neuron_data,
     int* d_total_number_of_spikes_stored_on_device,
     int* d_neuron_ids_of_stored_spikes_on_device,
     float* d_time_in_seconds_of_stored_spikes_on_device,
     int timestep_grouping,
     unsigned int current_time_in_timesteps,
     float timestep,
     size_t total_number_of_neurons){

      int idx = threadIdx.x + blockIdx.x * blockDim.x;
      int loc = (current_time_in_timesteps / timestep_grouping) % 2;
      while (idx < neuron_data->num_activated_neurons[loc]) {
        int i = atomicAdd(&d_total_number_of_spikes_stored_on_device[0], 1);
        d_neuron_ids_of_stored_spikes_on_device[i] = neuron_data->activated_neuron_ids[idx];
        d_time_in_seconds_of_stored_spikes_on_device[i] = (current_time_in_timesteps + neuron_data->activation_subtimesteps[idx])*timestep;
        
        idx += blockDim.x * gridDim.x;
      }
    }
  }
}
