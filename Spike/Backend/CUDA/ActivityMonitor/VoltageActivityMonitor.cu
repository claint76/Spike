// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/ActivityMonitor/VoltageActivityMonitor.hpp"

SPIKE_EXPORT_BACKEND_TYPE(CUDA, VoltageActivityMonitor);

namespace Backend {
  namespace CUDA {
    VoltageActivityMonitor::~VoltageActivityMonitor() {
    }

    void VoltageActivityMonitor::reset_state() {
      ActivityMonitor::reset_state();
      num_measurements = 0;
    }

    void VoltageActivityMonitor::prepare() {
      ActivityMonitor::prepare();
      neurons_frontend = frontend()->neurons;
      neurons_backend =
        dynamic_cast<::Backend::CUDA::LIFSpikingNeurons*>(neurons_frontend->backend());
    }

    void VoltageActivityMonitor::copy_data_to_host(){
      frontend()->neuron_measurements = (float*)realloc(frontend()->neuron_measurements, sizeof(float)*(frontend()->num_measurements + num_measurements));
      for (int i = 0; i < num_measurements; i++){
        frontend()->neuron_measurements[frontend()->num_measurements + i] = neuron_measurements[i];
      }
      frontend()->num_measurements += num_measurements;
      reset_state();
    }

    void VoltageActivityMonitor::collect_measurement
    (unsigned int current_time_in_timesteps, float timestep) {
      CudaSafeCall(hipMemcpy(frontend()->neuron_measurements + num_measurements,
                              neurons_backend->membrane_potentials_v + frontend()->neuron_id,
                              sizeof(float), 
                              hipMemcpyDeviceToHost));

      num_measurements++;

      if (num_measurements == max_num_measurements)
        copy_data_to_host();
    }


  }
}

